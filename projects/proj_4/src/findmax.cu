/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 To compile: 
    nvcc -arch=sm_60 reduction_kernel.cu
 To run with the array size 2^20, expo dist mean 5, and init seed 17:
    ./a.out 20 5 17
 */
#include <stdio.h>
#include <stdlib.h>
//#include <ldshr.h>

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<class T>
T reduceCPU(T *data, int size)
{
    T max = data[0];

    for (int i = 1; i < size; i++)
    {
        if (max < data[i])
            max = data[i];
    }

    return max;
}

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
__global__ void
reduce(double *g_idata, double *g_odata, unsigned int n)
{
    double *sdata = SharedMemory<double>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            //sdata[tid] += sdata[tid + s];
            if (sdata[0] < sdata[tid])
                sdata[0] = sdata[tid];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


// CUDA Runtime
#include <hip/hip_runtime.h>

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



//int main()  // for test findmax.cu
extern "C" double reduceMax(int nValue, int meanValue, int seedValue)
{
        int n = nValue; // 20;
        int mean = meanValue;  // 5;
        int seed = seedValue;  // 17
        
        int size = 1<<n;    // number of elements to reduce
        int maxThreads = 256;  // number of threads per block

        // create random input data on CPU
        unsigned int bytes = size * sizeof(double);

        double *h_idata = (double *) malloc(bytes);

        srand48(seed);
        for (int i=0; i<size; i++)
        {
                // h_idata[i] = 1.0; // for testing
                // expo dist with mean 5.0
                h_idata[i] = -mean * log(drand48());
        }
        //h_idata[5] = 1998.05;// for testing
        //h_idata[size - 1000] = 2019.04;   // for testing

        int numBlocks = size / maxThreads;
        int numThreads = size;

        int smemSize = maxThreads * sizeof(double);

        // allocate mem for the result on host side
        double *h_odata = (double *) malloc(numBlocks*sizeof(double));

        // allocate device memory and data
        double  *d_idata = NULL;
        double *d_odata = NULL;

        checkCudaErrors(hipMalloc((void **) &d_idata, bytes));
        checkCudaErrors(hipMalloc((void **) &d_odata, numBlocks*sizeof(double)));

        // copy data directly to device memory
        checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));

        reduce<<<numBlocks,maxThreads,smemSize>>>(d_idata, d_odata, numThreads);

        int s=numBlocks;

        while (s > 1) {
            reduce<<<(s+maxThreads-1)/maxThreads,maxThreads,smemSize>>>(d_odata, d_odata, s);
            s = (s+maxThreads-1)/maxThreads;
        }

        checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(double), hipMemcpyDeviceToHost));

        //printf("%f\n\n", h_odata[0]);     // this is not the final result

        checkCudaErrors(hipFree(d_idata));
        checkCudaErrors(hipFree(d_odata));

        // 2^20/2^8 = 2^12; 2^12 / 2^8 = 2^4; 
        // so the 2^4 number should be compared in CPU
        double cpu_result = reduceCPU<double>(h_idata, size);
   
        return cpu_result;
}
